#include "hip/hip_runtime.h"

#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "getopt.h"

// For CUDA printf
#include <stdio.h>

#ifndef TIME_TILE_SIZE
#warning TIME_TILE_SIZE is not set, defaulting to 1
#define TIME_TILE_SIZE 1
#endif

#ifndef FloatType
#define FloatType float
#endif

#if FloatType == float
#define ZERO 0.0f
#define THREE 3.0f
#else
#define ZERO 0.0
#define THREE 3.0
#endif

// Command-line options
const option commandLineOptions[] = {
  {"csv", no_argument, 0, 'e'},
  {"print", no_argument, 0, 'p'},
  {"seed", required_argument, 0, 'r'},
  {"size", required_argument, 0, 'n'},
  {"tilex", required_argument, 0, 'x'},
  {"time", required_argument, 0, 't'},
  {"verbose", no_argument, 0, 'v'},
  {"verify", no_argument, 0, 'c'},
  {0, 0, 0, 0}
};


#define CHECK_CALL(err)   checkCudaCall(err, __FILE__, __LINE__)
#define SYNC_DEVICE()     syncDevice(__FILE__, __LINE__)
#define ASSERT_STATE(msg) checkCudaState(msg, __FILE__, __LINE__)

inline void checkCudaCall(hipError_t   err,
                          const char *file,
                          const int   line) {
  if(hipSuccess != err) {
    std::cerr << file << "(" << line << ") :  checkCudaCall failed - " <<
      hipGetErrorString(err) << std::endl;
    //assert(false && "Cuda error");
    exit(-1);
  }
}

inline void syncDevice(const char *file,
                       const int   line) {
  //hipError_t err = hipDeviceSynchronize();
  //hipError_t err  = hipCtxSynchronize();
  //checkCudaCall(err, file, line);

  hipDeviceSynchronize();
}

inline void checkCudaState(const char *errorMessage,
                           const char *file,
                           const int   line) {
  hipError_t err = hipGetLastError();
  if(hipSuccess != err) {
    std::cerr << errorMessage << std::endl;
    checkCudaCall(err, file, line);
  }
}


// Print function
template <typename T>
void printValue(const char* name, T value, bool csv) {
  std::cout << name;
  std::cout << (csv ? "," : ":\t");
  std::cout << value << "\n";
}


// Kernels
__global__
void jacobi_1d_kernel_notimetiling(FloatType* input, FloatType* output,
                                   int32_t problemSize) {
  // Determine out start position
  int baseOffset = blockIdx.x * blockDim.x + threadIdx.x;

  // For no time tiling, just do everything in global memory
  FloatType c = input[baseOffset];
  FloatType l = (baseOffset > 0) ? input[baseOffset-1] : ZERO;
  FloatType r = (baseOffset < (problemSize-1)) ? input[baseOffset+1] : ZERO;

  FloatType average = (l + c + r) / THREE;

  output[baseOffset] = average;
}

extern __shared__ FloatType buffer[];

__global__
void jacobi_1d_kernel_overlapped(FloatType* input, FloatType* output,
                                 int32_t problemSize) {
  // Determine out start position
  int baseOffset = blockIdx.x * (blockDim.x-2*(TIME_TILE_SIZE-1)) + threadIdx.x;
  baseOffset -= TIME_TILE_SIZE-1;

  // Load data into shared
  buffer[threadIdx.x] = (baseOffset >= 0) ? ((baseOffset <= (problemSize-1)) ? input[baseOffset] : ZERO) : ZERO;
  __syncthreads();
  printf("[%d, %d]: Read input at %d (%f)\n", blockIdx.x, threadIdx.x, baseOffset, buffer[threadIdx.x]);

  // Perform the time iterations
#pragma unroll
  for(int t = 0; t < TIME_TILE_SIZE; ++t) {
    FloatType c = buffer[threadIdx.x];
    FloatType l = (threadIdx.x > 0) ? buffer[threadIdx.x-1] : ZERO;
    FloatType r = (threadIdx.x < (blockDim.x-1)) ? buffer[threadIdx.x+1] : ZERO;

    FloatType average = (l + c + r) / THREE;

    if(threadIdx.x == 0 && blockIdx.x == 1) {
      printf("f(%f, %f, %f) = %f\n", l, c, r, average);
    }

    // Sync before overwriting shared
    __syncthreads();

    buffer[threadIdx.x] = ((baseOffset >= 0) && (baseOffset <= (problemSize-1))) ? average : buffer[threadIdx.x];

    // Sync before re-reading shared
    __syncthreads();
  }

  if(threadIdx.x >= (TIME_TILE_SIZE-1) &&
     threadIdx.x <= (blockDim.x-1-(TIME_TILE_SIZE-1))) {
    output[baseOffset] = buffer[threadIdx.x];
    printf("[%d, %d]: Write output at %d\n", blockIdx.x, threadIdx.x, baseOffset);
  }
}

// Host function
void jacobi_1d_host(FloatType* input, FloatType* output,
                    int32_t problemSize, int32_t timeSteps) {
  FloatType* A = new FloatType[problemSize];
  FloatType* B = new FloatType[problemSize];

  memcpy(A, input, sizeof(FloatType) * problemSize);
  
  for(int t = 0; t < timeSteps; ++t) {
    for(int i = 0; i < problemSize; ++i) {
      FloatType c = A[i];
      FloatType l = (i > 0) ? A[i-1] : ZERO;
      FloatType r = (i < (problemSize-1)) ? A[i+1] : ZERO;

      FloatType average = (l + c + r) / THREE;

      if(i == 8) {
        printf("host f(%f, %f, %f) = %f\n", l, c, r, average);
      }
      
      B[i] = average;
    }
    std::swap(A, B);
  }
  
  for(int i = 0; i < problemSize; ++i) {
    output[i] = A[i];
  }

  delete [] A;
  delete [] B;
}

// Comparison function
void compareResults(FloatType* host, FloatType* device, int32_t problemSize,
                    bool csv) {
  double errorNorm, refNorm, diff;
  errorNorm = 0.0;
  refNorm = 0.0;

  for(int i = 0; i < problemSize; ++i) {
    diff = host[i] - device[i];
    errorNorm += diff * diff;
    refNorm += host[i] * host[i];
  }

  errorNorm = std::sqrt(errorNorm);
  refNorm = std::sqrt(refNorm);

  printValue("Error Norm", errorNorm, csv);
  printValue("Ref Norm", refNorm, csv);
  
  if(std::abs(refNorm) < 1e-7) {
    printValue("Correctness", "FAILED", csv);
  }
  else if((errorNorm / refNorm) > 1e-5) {
    printValue("Correctness", "FAILED", csv);
  }
  else {
    printValue("Correctness", "PASSED", csv);
  }
}

// Timer function
double rtclock(){
  timeval tp;
  gettimeofday(&tp, NULL);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}

// Program entry-point
int main(int argc, char** argv) {

  int32_t problemSize = 4096;
  int32_t timeSteps = 64;
  int32_t tileSizeX = 64;
  int32_t randomSeed = time(NULL);
  bool verbose = false;
  bool verify = false;
  bool print = false;
  bool csv = false;
  
  // Always use device 0
  hipSetDevice(0);

  // Parse options
  int opt;
  while((opt = getopt_long(argc, argv, "cen:pr:t:vx:", commandLineOptions, NULL)) != -1) {
    switch(opt) {
      case 'c':
        verify = true;
        break;
      case 'e':
        csv = true;
        break;
      case 'n':
        problemSize = atoi(optarg);
        break;
      case 'p':
        print = true;
        break;
      case 'r':
        randomSeed = atoi(optarg);
        break;
      case 't':
        timeSteps = atoi(optarg);
        break;
      case 'v':
        verbose = true;
        break;
      case 'x':
        tileSizeX = atoi(optarg);
        break;
      default:
        std::cerr << "[WARNING] Unknown option: " << opt << "\n";
        break;
    }
  }

  srand(randomSeed);

  // Print out experiment parameters
  if(verbose) {
    printValue("Problem Size", problemSize, csv);
    printValue("Time Steps", timeSteps, csv);
    printValue("Random Seed", randomSeed, csv);
  }

  // Allocate buffers
  FloatType* hostA;
  FloatType* hostB;
  FloatType* deviceA;
  FloatType* deviceB;
  FloatType* refOutput;
  
  // Allocate host buffers
  hostA = new FloatType[problemSize];
  hostB = new FloatType[problemSize];

  if(verbose) {
    printValue("Buffer Size", (problemSize * sizeof(FloatType)), csv);
  }

  // Allocate device buffers
  CHECK_CALL(hipMalloc((void**)&deviceA, sizeof(FloatType) * problemSize));
  CHECK_CALL(hipMalloc((void**)&deviceB, sizeof(FloatType) * problemSize));

  // Randomize the input
  for(int i = 0; i < problemSize; ++i) {
    hostA[i] = (FloatType)rand() / (FloatType)RAND_MAX;
    hostB[i] = ZERO;
  }

  // Compute reference
  if(verify) {
    refOutput = new FloatType[problemSize];
    jacobi_1d_host(hostA, refOutput, problemSize, timeSteps);
  }
  else {
    refOutput = NULL;
  }
  
  // Copy to device
  CHECK_CALL(hipMemcpy(deviceA, hostA, sizeof(FloatType) * problemSize,
                        hipMemcpyHostToDevice));
  CHECK_CALL(hipMemcpy(deviceB, hostB, sizeof(FloatType) * problemSize,
                        hipMemcpyHostToDevice));
  
  // Setup the kernel
  FloatType* input = deviceA;
  FloatType* output = deviceB;
  dim3 grid(problemSize / tileSizeX);
  dim3 block(tileSizeX + 2*(TIME_TILE_SIZE-1));

  if(verbose) {
    printValue("Block Size", block.x, csv);
    printValue("Grid Size", grid.x, csv);
  }
  
  // Run the kernel
  double startTime = rtclock();
  for(int t = 0; t < timeSteps; t += TIME_TILE_SIZE) {
    const int sharedMemSize = block.x * sizeof(FloatType);
#if TIME_TILE_SIZE == 1
    jacobi_1d_kernel_notimetiling<<<grid, block>>>(input, output, problemSize);
#else
    jacobi_1d_kernel_overlapped<<<grid, block, sharedMemSize>>>(input, output,
                                                                problemSize);
#endif
    std::swap(input, output);
  }
  SYNC_DEVICE();
  ASSERT_STATE("Kernel");
  double endTime = rtclock();
  double elapsedTime = endTime - startTime;
  
  printValue("Elapsed Time", elapsedTime, csv);

  double flops = problemSize * 3.0 * timeSteps;
  double gflops = flops / elapsedTime / 1e9;

  printValue("GFlop/s", gflops, csv);

  CHECK_CALL(hipMemcpy(hostB, input, sizeof(FloatType) * problemSize,
                        hipMemcpyDeviceToHost));
                          
  if(verify) {
    compareResults(refOutput, hostB, problemSize, csv);
  }

  if(print) {
    std::cout << "GPU -> CPU\n";
    for(int i = 0; i < problemSize; ++i) {
      std::cout << hostB[i];
      if(verify) {
        std::cout << " -> " << refOutput[i];
      }
      std::cout << "\n";
    }
  }
  
  // Free buffers
  delete [] hostA;
  delete [] hostB;
  if(refOutput)
    delete [] refOutput;
  CHECK_CALL(hipFree(deviceA));
  CHECK_CALL(hipFree(deviceB));
  
  return 0;
}
